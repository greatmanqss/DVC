#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/lgd_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void LgdLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, 
    const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count()/bottom[0]->num();
  const Dtype alpha = 4*top[0]->cpu_diff()[0] / bottom[0]->num()/(bottom[0]->num()-1)/(bottom[0]->num()-2)/(bottom[0]->num()+1);
  caffe_gpu_set(bottom[0]->count(),Dtype(0.),bottom[0]->mutable_gpu_diff());
  //caffe_gpu_set(diff_.count(),Dtype(0.),diff_.mutable_gpu_data());
  //caffe_gpu_set(realLoss_.count(),Dtype(0.),realLoss_.mutable_gpu_data());
  Dtype loss = 0.;
  const Dtype epsilon = 0.0001;
  const Dtype lamda = 1.0;
  for(int i = 0; i < bottom[0]->num(); i++) {
	  int offset_a = bottom[0]->offset(i);
	  for(int j = i+1; j < bottom[0]->num(); j++) {
		  int offset_b = bottom[0]->offset(j);
		  caffe_gpu_sub(
          count,
          bottom[0]->gpu_data()+offset_a,
          bottom[0]->gpu_data()+offset_b,
          diff1_.mutable_gpu_data());
		  
          Dtype dot1 = 0.;
		  caffe_gpu_dot(count, diff1_.gpu_data(), diff1_.gpu_data(),&dot1);
		  Dtype tmp1 = bottom[1]->cpu_data()[i*bottom[0]->num()+j]+epsilon;
		  dot1 = (dot1+lamda)/tmp1;
		  for (int m = i; m < bottom[0]->num();m++) {
			  int offset_c = bottom[0]->offset(m);
			  for (int n = m+1; n < bottom[0]->num();n++) {
				  if (m*bottom[0]->num()+n <= i*bottom[0]->num()+j)
					  continue;
				  int offset_d = bottom[0]->offset(n);
				  caffe_gpu_sub(
                  count,
                  bottom[0]->gpu_data()+offset_c,
                  bottom[0]->gpu_data()+offset_d,
                  diff2_.mutable_gpu_data());
                  Dtype dot2 = 0.;
				  Dtype tmp2 = bottom[1]->cpu_data()[m*bottom[0]->num()+n]+epsilon;
				  caffe_gpu_dot(count, diff2_.gpu_data(), diff2_.gpu_data(),&dot2);
		          dot2 = (dot2+lamda)/tmp2;
				  
				  Dtype diff_ij_mn = dot1 - dot2;
                  loss += abs(diff_ij_mn);	

                  Dtype sign = (diff_ij_mn >= 0) ? 1 : -1;
                  caffe_gpu_axpby(
			      count,
			      2*alpha*sign/tmp1,
			      diff1_.gpu_data(),
			      Dtype(1.),
			      bottom[0]->mutable_gpu_diff()+offset_a);				  
				  
				  caffe_gpu_axpby(
			      count,
			      -2*alpha*sign/tmp1,
			      diff1_.gpu_data(),
			      Dtype(1.),
			      bottom[0]->mutable_gpu_diff()+offset_b);
				  
				  caffe_gpu_axpby(
			      count,
			      -2*alpha*sign/tmp2,
			      diff2_.gpu_data(),
			      Dtype(1.),
			      bottom[0]->mutable_gpu_diff()+offset_c);
				  
				  caffe_gpu_axpby(
			      count,
			      2*alpha*sign/tmp2,
			      diff2_.gpu_data(),
			      Dtype(1.),
			      bottom[0]->mutable_gpu_diff()+offset_d);
			  }
		  }	
	  }
  }
  
  loss = loss * alpha;
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void LgdLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }	
}

INSTANTIATE_LAYER_GPU_FUNCS(LgdLossLayer);

}  // namespace caffe
