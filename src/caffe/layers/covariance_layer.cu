#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <algorithm>
#include <fstream>
#include "caffe/layers/covariance_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
 
template <typename Dtype>
__global__ void NonSingularize(const int nthreads, Dtype* top_data, const Dtype miu) {
  Dtype trace = 0;
  for(int i = 0; i < nthreads; i++){
     trace = trace + top_data[i*nthreads + i];
  } 
  CUDA_KERNEL_LOOP(index, nthreads) {
	top_data[index*nthreads + index] = top_data[index*nthreads + index] + miu*trace;
  }
}
template <typename Dtype>
__global__ void OuterProductBackward(const int nthreads, const Dtype* diff,
     Dtype* kron) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype val = diff[index];
	int offset_kron = index*nthreads*nthreads;
	
	for(int index1 = 0; index1 < nthreads; index1++) {
	  kron[offset_kron+index1*nthreads+index1] = val;// diagnal elements 
	  kron[offset_kron+index1*nthreads+index] = kron[offset_kron+index1*nthreads+index] + diff[index1];
	}
  }
}
template <typename Dtype>
__global__ void nonsingularizeBackward(const int nthreads, const Dtype* diff,
     Dtype* kron, const Dtype miu) {
  CUDA_KERNEL_LOOP(index, nthreads) {
	int offset_kron = index*(nthreads+1)*nthreads;
	
	for(int index1 = 0; index1 < nthreads; index1++) {
	  kron[offset_kron+index1] = kron[offset_kron+index1] + 2*miu*diff[index1];
	}
  }
}

template <typename Dtype>
void CovarianceLayer<Dtype>::Forward_gpu(
        const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
	//int batch_size = bottom[1]->num();//qss
    Dtype  first_label = bottom[1]->cpu_data()[1];//qss
    Dtype* seidx_data = start_end_idx_.mutable_cpu_data();
    Dtype* mean_data = mean_.mutable_gpu_data();
    Dtype* label_data = NULL;
    if(top.size()>1){
        label_data = top[1]->mutable_cpu_data();
        label_data[0] = bottom[1]->cpu_data()[0];
    }
    
    // dim of frame feature
    const int count = bottom[0]->channels()*bottom[0]->height()*bottom[0]->width();
    caffe_gpu_set(mean_.count(), Dtype(0.), mean_data);
    
	if(bottom.size()>2){
		caffe_copy(bottom[1]->count(),bottom[1]->cpu_data(),label_data);
		caffe_copy(bottom[2]->count(),bottom[2]->cpu_data(),seidx_data);
		for(int clip_idx = 1; clip_idx < video_num_; clip_idx++){
			int start = start_end_idx_.cpu_data()[clip_idx*2];
            int end = start_end_idx_.cpu_data()[clip_idx*2+1];
            Dtype coeff = 1.0 / (end - start);
            int offset_mean = mean_.offset(clip_idx);
            for (int frame_idx = start; frame_idx <= end; frame_idx++)
           {
              int offset_bottom = bottom[0]->offset(frame_idx);
              // fi - m --> differ_
              caffe_gpu_axpy(count, Dtype(1.0), bottom[0]->cpu_data()+offset_bottom, mean_data+offset_mean);
           }
		   caffe_gpu_scal(count, coeff, mean_data+offset_mean);
		}
	}
	else{
	    seidx_data[0] = 0;
        int clip_count = 1;
	
        int offset_mean = mean_.offset(clip_count-1);
        int offset_bottom = bottom[0]->offset(0);
        // sum each clip's frame features
        caffe_gpu_axpy(count, Dtype(1.0), bottom[0]->gpu_data()+offset_bottom, mean_data+offset_mean);
	
        // calaculate m = 1/n * sum (fi) for each clip
        for(int i = 1; i < bottom[1]->num(); i++)
       {
           seidx_data[clip_count*2-1] = i;
           if(bottom[1]->cpu_data()[i*2+1] != first_label)// the end of the previous clip and begin of a new clip qss
          {
            seidx_data[clip_count*2-1] = i-1;// fix the end index of previous clip
            Dtype coeff  = 1.0/(seidx_data[clip_count*2-1] -seidx_data[clip_count*2-2] + 1);
            // average frame features of previous clip
            caffe_gpu_scal(count, coeff, mean_data+offset_mean);
            if (clip_count < video_num_)// config the new clip
            {
                seidx_data[clip_count*2] = i;
                first_label = bottom[1]->cpu_data()[i*2+1];//qss
                if(top.size()>1) {
                    label_data[clip_count] = bottom[1]->cpu_data()[i*2];//qss
                }
                clip_count ++;
                offset_mean = mean_.offset(clip_count-1);
            }
            else// all clips have been found
            {
                break;
            }
          }
          offset_bottom = bottom[0]->offset(i);
          caffe_gpu_axpy(count, Dtype(1.0), bottom[0]->gpu_data()+offset_bottom, mean_data+offset_mean);
       }
	
       // all clips just fullfill this batch, no redundant frames
       if(seidx_data[clip_count*2-1] == bottom[1]->num() -1 ){
          Dtype coeff  = 1.0/(seidx_data[clip_count*2-1] -seidx_data[clip_count*2-2] + 1);
          caffe_gpu_scal(count, coeff, mean_data+offset_mean);
       }
	
	
	}
	
	
    
	

    // calculate C = 1/(n-1) * sum {(fi-m)*(fi-m)'} for each clip.
    Dtype* top_data = top[0]->mutable_gpu_data();
    caffe_gpu_set(top[0]->count(), Dtype(0.), top_data);
    // covariance for each clip
    for(int clip_id = 0; clip_id < video_num_; clip_id++)
    {
        int start = start_end_idx_.cpu_data()[clip_id*2];
        int end = start_end_idx_.cpu_data()[clip_id*2+1];
        Dtype coeff = 1.0 / (end - start);
        int offset_top = top[0]->offset(clip_id);
        int offset_mean = mean_.offset(clip_id);
        for (int frame_id = start; frame_id <= end; frame_id++)
        {
            int offset_bottom = bottom[0]->offset(frame_id);
            // fi - m --> differ_
            caffe_gpu_sub(count,bottom[0]->gpu_data()+offset_bottom, mean_data+offset_mean, differ_.mutable_gpu_data() );
            caffe_gpu_gemm<Dtype>(CblasNoTrans,CblasNoTrans, count,count,1, coeff, differ_.gpu_data(),differ_.gpu_data(),Dtype(1),top_data+offset_top);
        }
        // make sure the nonsingular property of the covariance, add a small eye matrix
        //for (int row = 0; row < count; row++){//可以并行加速
        //   caffe_gpu_add_scalar(1,Dtype(miu_),top_data+offset_top+row*count+row);
        //}
		
		NonSingularize<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,top_data+offset_top,miu_);
		
    }
}


template <typename Dtype>
void CovarianceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                          const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    
    for (int i = 0; i < bottom.size(); ++i) {
        if (propagate_down[i]) {
		    const int count = bottom[i]->channels()*bottom[i]->height()*bottom[i]->width();
            const Dtype* top_diff = top[i]->gpu_diff();
            Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
            caffe_gpu_set(bottom[i]->count(), Dtype(0.), bottom_diff);
            const Dtype* mean_data = mean_.gpu_data();
                for(int clip_id = 0; clip_id < video_num_; clip_id++)
                {
                    int start = start_end_idx_.cpu_data()[clip_id*2];
                    int end = start_end_idx_.cpu_data()[clip_id*2+1];
                    Dtype coeff = 1.0 / (end - start);
                    int offset_top = top[i]->offset(clip_id);
                    int offset_mean = mean_.offset(clip_id);
                    for (int frame_id = start; frame_id <= end; frame_id++) 
                    {
                        int offset_bottom = bottom[i]->offset(frame_id);
                        caffe_gpu_sub(count,bottom[i]->gpu_data()+offset_bottom, mean_data+offset_mean, differ_.mutable_gpu_data());

                        // calc the kronecker product kron(differ_',I) + kron(I, differ_')
                        caffe_gpu_set(kron_.count(),Dtype(0.),kron_.mutable_gpu_data());
						
						// need for speed
						OuterProductBackward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,differ_.gpu_data(),kron_.mutable_gpu_data());
					    nonsingularizeBackward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,differ_.gpu_data(),kron_.mutable_gpu_data(),miu_);

                        caffe_gpu_gemm<Dtype>(CblasNoTrans,CblasNoTrans,1,count,count*count,coeff,top_diff+offset_top,kron_.gpu_data(),Dtype(0.),bottom_diff+offset_bottom) ;
                    }
                }

        }
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(CovarianceLayer);

}  // namespace caffe
